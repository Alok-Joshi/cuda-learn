#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>

using namespace std;

__global__ void mat_mult(int *A, int *B, int *C,  int rowSize){

    int i = threadIdx.x;
    int j = threadIdx.y;

    int index = j + i*rowSize;

    for(int  k = 0; k<rowSize; k++){

        /* C[i][j] = A[i][k] * B[k][j]; */
        int a_index = i + k*rowSize;
        int b_index = k + j*rowSize;
        C[index] += B[b_index]*A[a_index];
    }

}

void print_matrix(int *mat, int N, int M){

    for(int i = 0; i<N; i++){
        for(int j = 0; j<M; j++){
            int index = j + i*M;
            cout<<mat[index]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;

}
int main(){

    int N = 3;
    int M = 3;
    int bytes = M*N*sizeof(int);
    int h_a[N][M] = { {1,2,3}, {1,2,3}, {1,2,3} };
    int h_b[N][M] = { {1,2,3}, {1,2,3}, {1,2,3} };
    int h_c[N][M] =  {0,0,0,0,0,0,0,0,0};


    //memory operations
    int *d_a,*d_b,*d_c;
    hipMalloc((void**) &d_a, bytes);
    hipMalloc((void**) &d_b, bytes);
    hipMalloc((void**) &d_c, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes,hipMemcpyHostToDevice);

    //kernel launch
    dim3 grids(1);
    dim3 threads(N, M);
    print_matrix((int*) h_c, N, M);
    mat_mult<<<grids, threads>>>(d_a, d_b, d_c,N);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    print_matrix((int*) h_c, N, M);

}
