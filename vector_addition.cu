
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
using namespace std;

__global__ void add(int *A, int *B, int *C){
	int index = threadIdx.x;
 	C[index] = A[index] + B[index];
}


void print_vector(int *vec, int N){

	for(int i = 0; i<N; i++){
		cout<<vec[i]<<" ";
	}
	cout<<endl;
}

int main(){

	int const N = 3;
	int A_host[] = {1,2,3};
	int B_host[] = {1,2,3};
	int C_host[] = {0,0,0};

	int *A_device, *B_device, *C_device;

	hipMalloc(&A_device, N*sizeof(int));
	hipMalloc(&B_device, N*sizeof(int));
	hipMalloc(&C_device, N*sizeof(int));

	hipMemcpy(A_device, A_host, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_device, B_host, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(C_device, C_host, N*sizeof(int), hipMemcpyHostToDevice);

	print_vector(C_host,N);
	add<<<1,N>>>(A_device,B_device,C_device);
	hipMemcpy(C_host, C_device, N*sizeof(int), hipMemcpyDeviceToHost);
	print_vector(C_host,N);

}
